#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

__global__
void matmul(double *X, double *Y, double* Z, int N)
{
    int i = 32*blockIdx.y + threadIdx.y; 
    int j = 32*blockIdx.x + threadIdx.x;
    if (i < N && j < N)
    {
        for (int k = 0; k < N; ++k)
        {
            Z[i*N+j] += X[i*N+k] * Y[j*N+k];
        }
    }
}

void cuda(double *X, double *Y, double *Z, int N)
{
    //Allocate GPU memory for given matrices X, Y, Z:
    double *Xc, *Yc, *Zc;

    assert(hipMalloc(&Xc, N*N*sizeof(double)) == hipSuccess);
    assert(hipMemcpy(Xc, X, N*N*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc(&Yc, N*N*sizeof(double)) == hipSuccess);
    assert(hipMemcpy(Yc, Y, N*N*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);

    assert(hipMalloc(&Zc, N*N*sizeof(double)) == hipSuccess);
    assert(hipMemcpy(Zc, Z, N*N*sizeof(double), hipMemcpyHostToDevice) == hipSuccess);

    //Organize CUDA threads as 32*32 which is the limit anyways:
    dim3 dimBlock(32, 32);
    //Rest goes into the blocks:
    dim3 dimGrid(N/32, N/32);

    matmul <<<dimGrid,dimBlock>>> (Xc, Yc, Zc, N);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) 
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    //Get back the Z matrix from GPU memory to CPU's:
    assert(hipMemcpy(Z, Zc, N*N*sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);

    hipFree(Xc); hipFree(Yc); hipFree(Zc);
    //Freeing the memory is a good practise in itself:
}
